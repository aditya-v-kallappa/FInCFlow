#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

namespace {

template <typename scalar_t>
__global__ void cinc_cuda_inverse_kernel(
    const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> input,
    const torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> kernel,
    torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> output) {
  //batch index
  const int n = blockIdx.y;
  // column index
  const int c = blockIdx.x * blockDim.x + threadIdx.x;

  // TODO: Compute inverse in a block here

}

} // namespace

std::vector<torch::Tensor> cinc_cuda_inverse(
    torch::Tensor input,
    torch::Tensor kernel,
    torch::Tensor output) {

  const auto batch_size = input.size(0);
  const auto state_size = input.size(1);

  const int threads = 1024;
  const dim3 blocks((state_size + threads - 1) / threads, batch_size);

  AT_DISPATCH_FLOATING_TYPES(input.type(), "cinc_inverse_cuda", ([&] {
    cinc_cuda_inverse_kernel<scalar_t><<<blocks, threads>>>(
        input.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
        kernel.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
        output.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>());
  }));

  return {output};
}
